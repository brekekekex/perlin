#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "lodepng.h"

#define CELL_X 600
#define CELL_Y 600
#define GRAIN 10

struct perlin_map{
	double *heights;
	unsigned int cells_x;
	unsigned int cells_y;
	unsigned int grain;
};

struct perlin_map *perlin_map_new(unsigned int c_x, unsigned int c_y, unsigned int g)
{
	struct perlin_map *p = (struct perlin_map *)malloc(sizeof(struct perlin_map));
	assert(p != NULL);
	p->cells_x = c_x;
	p->cells_y = c_y;
	p->grain = g;
	p->heights = (double *)malloc(sizeof(double) * (p->cells_x * p->grain) * (p->cells_y * p->grain));
	return p;
}

void perlin_map_destroy(struct perlin_map **p) 
{
	assert(*p != NULL);
	free((*p)->heights);
	(*p)->cells_x = 0;
	(*p)->cells_y = 0;
	(*p)->grain = 0;
	free(*p);
	(*p) = NULL;
}

unsigned char *render_grayscale(const struct perlin_map *p)
{
	assert(p != NULL);
	unsigned char *im = (unsigned char *)malloc(sizeof(char) * (p->cells_x * p->grain) * (p->cells_y * p->grain) * 4);
	assert(im != NULL);
	// find min and max heights
	double min, max;
	min = (p->heights)[0];
	max = (p->heights)[0];
	for (int i = 0; i < (p->cells_x * p->grain); i++) {
		for (int j = 0; j < (p->cells_y * p->grain); j++) {
			if ((p->heights)[i + j * (p->cells_x * p->grain)] < min) {
				min = (p->heights)[i + j * (p->cells_x * p->grain)];
			}
			if ((p->heights)[i + j * (p->cells_x * p->grain)] > max) {
				max = (p->heights)[i + j * (p->cells_x * p->grain)];
			}
		}
	} 
	// write rgba (use alpha to encode normalized heights)
	for (int i = 0; i < (p->cells_x * p->grain); i++) {
		for (int j = 0; j < (p->cells_y * p->grain); j++) {
			im[4 * i + (4 * j * (p->cells_x * p->grain)) + 0] = 0;
			im[4 * i + (4 * j * (p->cells_x * p->grain)) + 1] = 0;
			im[4 * i + (4 * j * (p->cells_x * p->grain)) + 2] = 0;
			im[4 * i + (4 * j * (p->cells_x * p->grain)) + 3] = (char)(255. * (((p->heights)[i + j * (p->cells_x * p->grain)] - min) / (max - min)));
		}
	}
	return im;
}

__device__
double fade(int d)
{
	return (double)(6 * (d * d * d * d * d) - 15 * (d * d * d * d) + 10 * (d * d * d));
}

__device__
double linterp(double t, double a, double b)
{
	return a + t * (b - a);

}

__device__
double grad(int hash, int x_d, int y_d)
{
	switch (hash & 0x7) {
		case 0x0: return (double)(x_d + y_d);
		case 0x1: return (double)(-x_d + y_d);
		case 0x2: return (double)(x_d - y_d);
		case 0x3: return (double)(-x_d - y_d);
		case 0x4: return (double)(x_d);
		case 0x5: return (double)(-x_d);
		case 0x6: return (double)(y_d);
		case 0x7: return (double)(-y_d);
	}
}

__global__
void perlin_fill_heights(double *height_map, unsigned int c_x, unsigned int c_y, unsigned int g)
{
	int num_elems = (c_x * g) * (c_y * g); 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;


	// grid-stride loop
	for (int i = index; i < num_elems; i += stride) {
		int x_pos, y_pos;
		int x_disp, y_disp;
		double x_fade, y_fade;
		// un-linearise
		x_pos = i % (c_x * g);
		y_pos = (i - x_pos) / (c_x * g);
		// displacements
		x_disp = x_pos % g;
		y_disp = y_pos % g;
		// fade
		x_fade = fade(x_disp);
		y_fade = fade(y_disp);
		
	}
}

int main(void)
{
	// allocate host memory
	struct perlin_map *p = perlin_map_new(CELL_X, CELL_Y, GRAIN);

	for (int i = 0; i < (p->cells_x * p->grain); i++) {
		for (int j = 0; j < (p->cells_y * p->grain); j++) {	
			(p->heights)[i + j * (p->cells_x * p->grain)] = i + j;
		}
	} 	
	/*
	unsigned char *render = render_grayscale(p);
	unsigned char *png = 0;
	size_t pngsize;	
	unsigned int err = lodepng_encode32(&png, &pngsize, render, p->cells_x * p->grain, p->cells_y * p->grain);	
	lodepng_save_file(png, pngsize, "test.png");
	
*/	
	// allocate device memory
	//double *d_heights;
	//hipMalloc((void **)&d_heights, sizeof(double) * (CELL_X * GRAIN) * (CELL_Y * GRAIN)); 	
	perlin_map_destroy(&p);



	return 0;
}


